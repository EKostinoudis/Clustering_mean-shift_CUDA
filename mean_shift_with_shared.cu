#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define N 600                       //total points
#define D 2                         //dimensions
#define STANDARD_DEVIATION 1        //standard deviation
#define EPSILON 1e-4                //converge critirion

#define POINTS_FILE "x.bin"         //name of the file with the points




__global__ void meanShift(double *x0, double *y) {
    int index_y = blockIdx.x*D;       //y index
    int index_x = threadIdx.x*D;      //x index

    int i;
    double y_new[D], y_prev[D], m[D], m_norm;
    __shared__ double x[N*D];           //use shared memory for the x matrix
    __shared__ double denominator[N];   //variable to store the denominator sum
    __shared__ double numerator[N*D];   //variable to store the numerator sum



    for(int i=0;i<D;i++) {
        //init x matrix
        x[index_x + i] = x0[index_x + i];
    }


    __syncthreads();


    for(int i=0;i<D;i++) {
        //init y_prev
        y_prev[i] = x[index_y + i];
    }


    do{
        //do the subtraction
        for(i=0;i<D;i++) {
            y_new[i] = y_prev[i] - x[index_x + i];
        }


        //calculate norm
        denominator[threadIdx.x] = norm(D, y_new);

        if(denominator[threadIdx.x] <= STANDARD_DEVIATION) {
            //take the square
            denominator[threadIdx.x] *= denominator[threadIdx.x];

            //calculate the Gaussian kernel
            denominator[threadIdx.x] = exp(-denominator[threadIdx.x] / (2*STANDARD_DEVIATION));

            //calculate the numerator (for every j)
            for(i=0;i<D;i++) {
                numerator[index_x + i] = denominator[threadIdx.x] * x[index_x + i];
            }
        }
        else {
            denominator[threadIdx.x] = 0;
            for(i=0;i<D;i++) {
                numerator[index_x + i] = 0;
            }
        }


        __syncthreads();
        //reduction || do the sums
        for (unsigned int s=1;s<N;s*=2) {
            int index = 2 * s * threadIdx.x;
            if (index < N && (index + s) < N) {
                //denominator sum
                denominator[index] += denominator[index + s];

                //numerator sum
                for(i=0;i<D;i++) {
                    numerator[index*D + i] += numerator[(index + s)*D + i];
                }
            }
            __syncthreads();
        }


        for(int i=0;i<D;i++) {
            //calculate new y (y^k+1)
            y_new[i] = numerator[i] / denominator[0];

            //calculate m
            m[i] = y_new[i] - y_prev[i];

            //init y_prev for the next iteration
            y_prev[i] = y_new[i];
        }

        //calculate norm(m)
        m_norm = norm(D, m);

    } while(m_norm >= EPSILON);


    //return the final y
    for(i=0;i<D;i++) {
        y[index_y + i] = y_new[i];
    }

}



////////////////////////////////////////////////////////////////////////////////
//////////////////////////////// Main function /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {
    double *x, *y, *dev_x, *dev_y;
    int read;
    FILE *points, *result;
    hipError_t error;



    //allocate memory for the x matrix
    x = (double*) malloc(N*D*sizeof(double));
    if(x == NULL) {
        fprintf(stderr, "malloc fail.\n");
        exit(1);
    }


    //allocate memory for the y matrix || result
    y = (double*) malloc(N*D*sizeof(double));
    if(y == NULL) {
        fprintf(stderr, "malloc fail.\n");
        exit(1);
    }


    //open the points file
    points = fopen(POINTS_FILE,"rb");
    if(points == NULL) {
      fprintf(stderr, "Unable to open file\n");
      exit(1);
    }


    //read the points data
    read = fread(x,sizeof(double),N*D,points);
    if (read != N*D) {
        fprintf(stderr, "Unable to read data\n");
        exit(1);
    }


    //allocate device memory for the x matrix
    if(hipSuccess != hipMalloc((void**)&dev_x, N*D*sizeof(double))) {
        fprintf(stderr, "hipMalloc fail.\n");
        exit(1);
    }


    //allocate device memory for the y matrix
    if(hipSuccess != hipMalloc((void**)&dev_y, N*D*sizeof(double))) {
        fprintf(stderr, "hipMalloc fail.\n");
        exit(1);
    }


    //move x matrix to the device
    error = hipMemcpy(dev_x, x, N*D*sizeof(double), hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        fprintf(stderr, "Moving data to device fail.\n");
        exit(1);
    }


    //variables for time measurement
    struct timeval tval_before, tval_after, tval_result;


    //blocksize and thread size
    dim3 dimGrid(N);
    dim3 dimBlock(N);


    //start time measurement
    gettimeofday(&tval_before, NULL);


    //calculate the mean shift
    meanShift<<<dimGrid, dimBlock>>>(dev_x, dev_y);


    //wait for meanShift to complete
    if (hipSuccess != hipDeviceSynchronize()) {
        fprintf(stderr, "Fail synchronize\n");
        exit(1);
    }


    //end time measurement
    gettimeofday (&tval_after, NULL);


    //calculate the execute time
    timersub(&tval_after, &tval_before, &tval_result);


    //time in seconds
    //printf("Took %ld.%06ld sec\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

    //time in micro seconds
    printf("Took %ld micro seconds\n", (long int) ((tval_result.tv_sec) * 1.0e6) + tval_result.tv_usec);



    //take the y matrix values from the device
    error = hipMemcpy(y, dev_y, N*D*sizeof(double), hipMemcpyDeviceToHost);
    if(error != hipSuccess) {
        fprintf(stderr, "Moving data to device fail.\n");
        exit(1);
    }


    result = fopen("result.txt", "a");


    for(int i=0;i<N;i++) {
        for(int j=0;j<D;j++) {
            fprintf(result, "%lf ", y[i*D + j]);
        }
        fprintf(result, "\n");
    }


    free(x);
    free(y);
    hipFree(dev_x);
    hipFree(dev_y);
    fclose(points);
    fclose(result);
}
